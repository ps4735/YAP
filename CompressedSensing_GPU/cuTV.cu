#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <helper_functions.h>
#include <hip/hip_runtime_api.h>
#include <cusp/complex.h>
#include <thrust/transform.h>
#include <thrust/device_vector.h>
#include <thrust/functional.h>
#include <arrayfire.h>
#include <cstdio>
#include <cstdlib>

using namespace cusp;
using namespace thrust;

const unsigned int TILE_SIZE = 16; //32��16��
const unsigned int BLOCK_ROWS_SIZE = 16;//8��4��
const int KERNEL_LENGTH = 4;

__global__ void tv_vertical(cusp::complex<float>* d_A, cusp::complex<float>* d_B, unsigned int imageW, unsigned int imageH)  //d_BΪ�������
{
	__shared__ cusp::complex<float> tile1[TILE_SIZE][TILE_SIZE + 1];
	__shared__ cusp::complex<float> tile2[TILE_SIZE][TILE_SIZE + 1];
	int x = blockIdx.x * TILE_SIZE + threadIdx.x;
	int y = blockIdx.y * TILE_SIZE + threadIdx.y;
	tile1[threadIdx.y][threadIdx.x] = d_A[y*imageW + x];
	if (y < imageH - 1)
	{
		tile2[threadIdx.y][threadIdx.x] = d_A[(y + 1)*imageW + x];
	}
	__syncthreads();

	if (y < imageH - 1)
	{
		d_B[y*imageW + x] = tile2[threadIdx.y][threadIdx.x] - tile1[threadIdx.y][threadIdx.x];
	}
	if (y == imageH - 1)
		d_B[y*imageW + x] = 0.0;
}



__global__ void tv_horizontal(cusp::complex<float>* d_A, cusp::complex<float>* d_B, unsigned int imageW, unsigned int imageH)  //d_BΪ�������
{
	__shared__ cusp::complex<float> tile1[TILE_SIZE][TILE_SIZE + 1];
	__shared__ cusp::complex<float> tile2[TILE_SIZE][TILE_SIZE + 1];
	int x = blockIdx.x * TILE_SIZE + threadIdx.x;
	int y = blockIdx.y * TILE_SIZE + threadIdx.y;

	tile1[threadIdx.y][threadIdx.x] = d_A[y*imageW + x];
	if (x < imageH - 1)
	{
		tile2[threadIdx.y][threadIdx.x] = d_A[y*imageW + x + 1];
	}
	__syncthreads();

	if (x < imageH - 1)
	{
		d_B[y*imageW + x] = tile2[threadIdx.y][threadIdx.x] - tile1[threadIdx.y][threadIdx.x];
	}
	if (x == imageH - 1)
	{
		d_B[y*imageW + imageH - 1] = 0.0;
	}
}

__global__ void itv(cusp::complex<float>* d_A, cusp::complex<float>* d_B, cusp::complex<float>* d_C, unsigned int imageW, unsigned int imageH)   //d_CΪ�������
{
	__shared__ cusp::complex<float> tile1[TILE_SIZE][TILE_SIZE + 1];
	__shared__ cusp::complex<float> tile2[TILE_SIZE][TILE_SIZE + 1];
	int x = blockIdx.x * TILE_SIZE + threadIdx.x;
	int y = blockIdx.y * TILE_SIZE + threadIdx.y;

	if ((y > 0 && y < imageH - 1) && (x > 0 && x < imageW - 1))
	{
		tile1[threadIdx.y][threadIdx.x] = d_A[(y - 1)*imageW + x] - d_A[y*imageW + x];
		tile2[threadIdx.y][threadIdx.x] = d_B[y*imageW + x - 1] - d_B[y*imageW + x];
	}
	if (y == 0 || x == 0)
	{
		tile1[threadIdx.y][threadIdx.x] = 0.0;
		tile2[threadIdx.y][threadIdx.x] = 0.0;
	}
	if ((y == imageH - 1) && (x == imageW - 1))
	{
		tile1[threadIdx.y][threadIdx.x] = d_A[(y - 1)*imageW + x];
		tile2[threadIdx.y][threadIdx.x] = d_B[y*imageW + x - 1];
	}

	__syncthreads();

	d_C[y*imageW + x] = tile1[threadIdx.y][threadIdx.x] + tile2[threadIdx.y][threadIdx.x];
}

extern "C" void FTV(complex<float> * d_A, complex<float> * d_B, complex<float> * d_C, unsigned int imageW, unsigned int imageH)
{

//	size_t lpitch;
//	complex<float> *d_A, *d_B, *d_C;
//	checkCudaErrors(hipMallocPitch((void **)&d_A, &lpitch, imageW*sizeof(complex<float>), imageH));
//	checkCudaErrors(hipMallocPitch((void **)&d_B, &lpitch, imageW*sizeof(complex<float>), imageH));
//	checkCudaErrors(hipMallocPitch((void **)&d_C, &lpitch, imageW*sizeof(complex<float>), imageH));
//	checkCudaErrors(hipMemcpy2D(d_A, lpitch, af_d_A, sizeof(float)*2*imageW, sizeof(float)*2*imageW, imageH, hipMemcpyDeviceToDevice));

	dim3 dimBlock(TILE_SIZE, BLOCK_ROWS_SIZE, 1);
	dim3 dimGrid(imageW / TILE_SIZE, imageH / TILE_SIZE, 1);
	tv_vertical << <dimGrid, dimBlock>> >(d_A, d_B, imageW, imageH);
	tv_horizontal << <dimGrid, dimBlock>> >(d_A, d_C, imageW, imageH);

//	checkCudaErrors(hipMemcpy2D(af_d_B, lpitch, d_B, sizeof(float)*2*imageW, sizeof(float)*2*imageW, imageH, hipMemcpyDeviceToDevice));
//	checkCudaErrors(hipMemcpy2D(af_d_C, lpitch, d_C, sizeof(float) * 2 * imageW, sizeof(float) * 2 * imageW, imageH, hipMemcpyDeviceToDevice));
//	checkCudaErrors(hipFree(d_A));
//	checkCudaErrors(hipFree(d_B));
//	checkCudaErrors(hipFree(d_C));

}

extern "C" void IFTV(complex<float> *d_A, complex<float> * d_B, complex<float> * d_C, unsigned int imageW, unsigned int imageH) //d_CΪ�������
{

//	size_t lpitch;
//	complex<float> *d_A, *d_B, *d_C;
//	checkCudaErrors(hipMallocPitch((void **)&d_A, &lpitch, imageW*sizeof(complex<float>), imageH));
//	checkCudaErrors(hipMallocPitch((void **)&d_B, &lpitch, imageW*sizeof(complex<float>), imageH));
//	checkCudaErrors(hipMallocPitch((void **)&d_C, &lpitch, imageW*sizeof(complex<float>), imageH));
//	checkCudaErrors(hipMemcpy2D(d_A, lpitch, af_d_A, sizeof(float) * 2 * imageW, sizeof(float) * 2 * imageW, imageH, hipMemcpyDeviceToDevice));
//	checkCudaErrors(hipMemcpy2D(d_B, lpitch, af_d_B, sizeof(float) * 2 * imageW, sizeof(float) * 2 * imageW, imageH, hipMemcpyDeviceToDevice));

	dim3 dimBlock(TILE_SIZE, BLOCK_ROWS_SIZE, 1);
	dim3 dimGrid(imageW / TILE_SIZE, imageW / TILE_SIZE, 1);
	itv << <dimGrid, dimBlock >> >(d_A, d_B, d_C, imageW, imageH);

//	checkCudaErrors(hipMemcpy2D(af_d_C, lpitch, d_C, sizeof(float) * 2 * imageW, sizeof(float) * 2 * imageW, imageH, hipMemcpyDeviceToDevice));
//	checkCudaErrors(hipFree(d_A));
//	checkCudaErrors(hipFree(d_B));
//	checkCudaErrors(hipFree(d_C));
}